#include <iostream>
#include <vector>

#include "hip/hip_runtime.h"
#include "game_state.h"

int main() {
    // Initialize game state
    GameState gameState;
    gameState.init();

    // Main game loop
    while (!gameState.isGameOver()) {
        gameState.displayBoard();
        gameState.playTurn();
    }

    gameState.displayResults();
    return 0;
}
